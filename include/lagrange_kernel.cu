/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.

When using this code in a scientific project, please cite one or all of the
following papers:
*  Daniel Ruijters and Philippe Th�venaz,
   GPU Prefilter for Accurate Cubic B-Spline Interpolation, 
   The Computer Journal, vol. 55, no. 1, pp. 15-20, January 2012.
   http://dannyruijters.nl/docs/cudaPrefilter3.pdf
*  Daniel Ruijters, Bart M. ter Haar Romeny, and Paul Suetens,
   Efficient GPU-Based Texture Interpolation using Uniform B-Splines,
   Journal of Graphics Tools, vol. 13, no. 4, pp. 61-69, 2008.
\*--------------------------------------------------------------------------*/

#ifndef _CUDA_LAGRANGE_H_
#define _CUDA_LAGRANGE_H_

#define one_six ((float)0.166666666666)
#define m_one_six ((float)-0.166666666666)
//#include "cutil_math_bugfixes.h"
#include "cuda_helper_math.h"
#include "math_func.cu"

// Inline calculation of the lagrange convolution weights, without conditional statements
template<class T> inline __device__ void lagrange_weights(T fraction, T& w0, T& w1, T& w2, T& w3)
{
    const T x = fraction + 1.0f;
    const T xm1 = fraction;
    const T xm2 = x - 2.0f;
    const T xm3 = x - 3.0f;
    const T xm23 = xm2*xm3;
    const T xxm1 = x*xm1;

	w0 =  m_one_six * xm1 * xm23;
	w1 = 0.5f * x * xm23;
	w2 = -0.5f * xxm1 * xm3;
	w3 = one_six * xxm1 * xm2;
}

#endif // _CUDA_LAGRANGE_H_
