#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.

When using this code in a scientific project, please cite one or all of the
following papers:
*  Daniel Ruijters and Philippe Th�venaz,
   GPU Prefilter for Accurate Cubic B-Spline Interpolation, 
   The Computer Journal, vol. 55, no. 1, pp. 15-20, January 2012.
   http://dannyruijters.nl/docs/cudaPrefilter3.pdf
*  Daniel Ruijters, Bart M. ter Haar Romeny, and Paul Suetens,
   Efficient GPU-Based Texture Interpolation using Uniform B-Splines,
   Journal of Graphics Tools, vol. 13, no. 4, pp. 61-69, 2008.
\*--------------------------------------------------------------------------*/

#ifndef _3D_CUBIC_BSPLINE_PREFILTER_H_
#define _3D_CUBIC_BSPLINE_PREFILTER_H_

#include <stdio.h>
#include<hip/hip_runtime.h>
#include "cubicPrefilter_kernel.cu"

//--------------------------------------------------------------------------
// Global CUDA procedures
//--------------------------------------------------------------------------
template<class floatN>
__global__ void SamplesToCoefficients3DX(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in x-direction
	const uint y = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = (z * height + y) * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr, width, sizeof(floatN));
}

template<class floatN>
__global__ void SamplesToCoefficients3DY(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in y-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = z * height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr + x, height, pitch);
}

template<class floatN>
__global__ void SamplesToCoefficients3DZ(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in z-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = y * pitch;
	const uint slice = height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients(ptr + x, depth, slice);
}

//--------------------------------------------------------------------------
// Exported functions
//--------------------------------------------------------------------------

//! Convert the voxel values into cubic b-spline coefficients
//! @param volume  pointer to the voxel volume in GPU (device) memory
//! @param pitch   width in bytes (including padding bytes)
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
template<class floatN>
extern void CubicBSplinePrefilter3D(floatN* volume, uint pitch, uint width, uint height, uint depth)
{
	// Try to determine the optimal block dimensions
	uint dimX = min(min(PowTwoDivider(width), PowTwoDivider(height)), 64);
	uint dimY = min(min(PowTwoDivider(depth), PowTwoDivider(height)), 512/dimX);
	dim3 dimBlock(dimX, dimY);

	// Replace the voxel values by the b-spline coefficients
	dim3 dimGridX(height / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DX<floatN><<<dimGridX, dimBlock>>>(volume, pitch, width, height, depth);
	

	dim3 dimGridY(width / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DY<floatN><<<dimGridY, dimBlock>>>(volume, pitch, width, height, depth);


	dim3 dimGridZ(width / dimBlock.x, height / dimBlock.y);
	SamplesToCoefficients3DZ<floatN><<<dimGridZ, dimBlock>>>(volume, pitch, width, height, depth);

}

//! Convert the voxel values into cubic b-spline coefficients
//! @param volume  pointer to the voxel volume in GPU (device) memory
//! @param pitch   width in bytes (including padding bytes)
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
//! @note Prints stopwatch feedback
template<class floatN>
extern void CubicBSplinePrefilter3DTimer(floatN* volume, uint pitch, uint width, uint height, uint depth)
{

	// Try to determine the optimal block dimensions
	uint dimX = min(min(PowTwoDivider(width), PowTwoDivider(height)), 64);
	uint dimY = min(min(PowTwoDivider(depth), PowTwoDivider(height)), 512/dimX);
	dim3 dimBlock(dimX, dimY);

	// Replace the voxel values by the b-spline coefficients
	dim3 dimGridX(height / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DX<floatN><<<dimGridX, dimBlock>>>(volume, pitch, width, height, depth);



	dim3 dimGridY(width / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DY<floatN><<<dimGridY, dimBlock>>>(volume, pitch, width, height, depth);



	dim3 dimGridZ(width / dimBlock.x, height / dimBlock.y);
	SamplesToCoefficients3DZ<floatN><<<dimGridZ, dimBlock>>>(volume, pitch, width, height, depth);


}

/**************** PERIODIC Implementation for computing spline coefficients *****************************/
//--------------------------------------------------------------------------
// Global CUDA procedures
//--------------------------------------------------------------------------
template<class floatN>
__global__ void SamplesToCoefficients3DX_Periodic(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in x-direction
	const uint y = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = (z * height + y) * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients_periodic(ptr, width, sizeof(floatN));
}

template<class floatN>
__global__ void SamplesToCoefficients3DY_Periodic(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in y-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint z = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = z * height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients_periodic(ptr + x, height, pitch);
}

template<class floatN>
__global__ void SamplesToCoefficients3DZ_Periodic(
	floatN* volume,		// in-place processing
	uint pitch,			// width in bytes
	uint width,			// width of the volume
	uint height,		// height of the volume
	uint depth)			// depth of the volume
{
	// process lines in z-direction
	const uint x = blockIdx.x * blockDim.x + threadIdx.x;
	const uint y = blockIdx.y * blockDim.y + threadIdx.y;
	const uint startIdx = y * pitch;
	const uint slice = height * pitch;

	floatN* ptr = (floatN*)((uchar*)volume + startIdx);
	ConvertToInterpolationCoefficients_periodic(ptr + x, depth, slice);
}

//--------------------------------------------------------------------------
// Exported functions
//--------------------------------------------------------------------------

//! Convert the voxel values into cubic b-spline coefficients
//! @param volume  pointer to the voxel volume in GPU (device) memory
//! @param pitch   width in bytes (including padding bytes)
//! @param width   volume width in number of voxels
//! @param height  volume height in number of voxels
//! @param depth   volume depth in number of voxels
template<class floatN>
extern void CubicBSplinePrefilter3D_Periodic(floatN* volume, uint pitch, uint width, uint height, uint depth)
{
	// Try to determine the optimal block dimensions
    float time=0, dummy_time=0;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    
    hipEventRecord(startEvent,0); 

	uint dimX = min(min(PowTwoDivider(width), PowTwoDivider(height)), 64);
	uint dimY = min(min(PowTwoDivider(depth), PowTwoDivider(height)), 512/dimX);
	dim3 dimBlock(dimX, dimY);

	// Replace the voxel values by the b-spline coefficients
	dim3 dimGridX(height / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DX_Periodic<floatN><<<dimGridX, dimBlock>>>(volume, pitch, width, height, depth);
    if ( hipSuccess != hipGetLastError())
        printf("Error in running the interp3D kernel\n");
	
	dim3 dimGridY(width / dimBlock.x, depth / dimBlock.y);
	SamplesToCoefficients3DY_Periodic<floatN><<<dimGridY, dimBlock>>>(volume, pitch, width, height, depth);
    if ( hipSuccess != hipGetLastError())
        printf("Error in running the interp3D kernel\n");

    dim3 dimGridZ(width / dimBlock.x, height / dimBlock.y);
	SamplesToCoefficients3DZ_Periodic<floatN><<<dimGridZ, dimBlock>>>(volume, pitch, width, height, depth);
    if ( hipSuccess != hipGetLastError())
        printf("Error in running the interp3D kernel\n");

    hipEventRecord(stopEvent,0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dummy_time, startEvent, stopEvent);
    time+=dummy_time/1000;
    hipDeviceSynchronize();
    
    printf("\n Computation of Spline coefficients for 3D data on grid N=%dx%dx%d took %0.2E sec\n\n", width, height, depth, time);
    
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}




#endif  //_3D_CUBIC_BSPLINE_PREFILTER_H_
