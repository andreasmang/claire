#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2016.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE.  If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#ifndef _DIFFERENTIATIONKERNEL_CPP_
#define _DIFFERENTIATIONKERNEL_CPP_

#include "DifferentiationKernel.hpp"
#include "cuda_helper.hpp"

#include "DifferentiationKernel.txx"

template<typename KernelFn, typename ... Args>
__global__ void SpectralKernelGPU(int3 wave, int3 nx, int3 nl, Args ... args) {
  int i1 = threadIdx.x + blockIdx.x*blockDim.x;
  int i2 = blockIdx.y;
  int i3 = blockIdx.z;
  
  if (i1 < nl.x) {
    wave.x += i1;
    wave.y += i2;
    wave.z += i3;

    ComputeWaveNumber(wave, nx);
    int i = GetLinearIndex(i1, i2, i3, nl);

    KernelFn::call(i, wave, args...);
  }
}
template<typename KernelFn, typename ... Args>
PetscErrorCode SpectralKernelCallGPU(IntType nstart[3], IntType nx[3], IntType nl[3], 
    Args ... args) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  dim3 block(256,1,1);
  dim3 grid((nl[0] + 255)/256,nl[1],nl[2]);
  int3 wave, nx3, nl3;
  wave.x = nstart[0]; wave.y = nstart[1]; wave.z = nstart[2];
  nx3.x = nx[0]; nx3.y = nx[1]; nx3.z = nx[2];
  nl3.x = nl[0]; nl3.y = nl[1]; nl3.z = nl[2];
  
  if (nl[0]*nl[1]*nl[2] > 0) {
    SpectralKernelGPU<KernelFn><<<grid, block>>>(wave, nx3, nl3, args...);
    ierr = hipDeviceSynchronize(); CHKERRCUDA(ierr);
    ierr = cudaCheckKernelError(); CHKERRCUDA(ierr);
  }
  
  PetscFunctionReturn(ierr);
}

namespace reg {
namespace DifferentiationKernel {
  
PetscErrorCode VectorField::Laplacian(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::LaplacianTol(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ScalarType lognx = 0.;
  lognx += log2(static_cast<ScalarType>(nx[0]));
  lognx += log2(static_cast<ScalarType>(nx[1]));
  lognx += log2(static_cast<ScalarType>(nx[2]));
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianFilterKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, tol*lognx); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::Bilaplacian(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<2> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<2> >(nstart, nx, nl,
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::Trilaplacian(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<3> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<3> >(nstart, nx, nl,
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::TrilaplacianFunctional(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = ThrowError("trilaplacian operator not implemented"); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::InverseLaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianSqrtKernel<1> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::InverseBilaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      /// scale/sqrt(b0*|lapik|^2) = scale/(sqrt(b0)*|lapik|)
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, sqrt(b0)); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::InverseTrilaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianSqrtKernel<3> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, sqrt(b0)); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode VectorField::Leray(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<LerayKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    scale, b0, b1); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

} // namespace DifferentiationKernel
} // namespace reg

#endif
