#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2016.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE.  If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#ifndef _DIFFERENTIATIONKERNEL_CPP_
#define _DIFFERENTIATIONKERNEL_CPP_

#include "DifferentiationKernel.hpp"
#include "cuda_helper.hpp"

#include "DifferentiationKernel.txx"

using KernelUtils::SpectralKernelCallGPU;

namespace reg {

PetscErrorCode DifferentiationKernel::ScalarLaplacian(ScalarType b0) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<NLaplacianKernel<1> >(nstart, nx, nl, 
    pXHat[0], b0*scale); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}


PetscErrorCode DifferentiationKernel::LaplacianMod(ScalarType b0) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<NLaplacianModKernel<1> >(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    scale, b0); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}
PetscErrorCode DifferentiationKernel::Laplacian(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::LaplacianTol(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ScalarType lognx = 0.;
  lognx += log2(static_cast<ScalarType>(nx[0]));
  lognx += log2(static_cast<ScalarType>(nx[1]));
  lognx += log2(static_cast<ScalarType>(nx[2]));
  
  KernelUtils::array3_t<ComplexType*> v;
  v.x = pXHat[0];
  v.y = pXHat[1];
  v.z = pXHat[2];
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianFilterKernel<1> >(nstart, nx, nl, v, 
      b0*scale, tol*lognx); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<1> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Bilaplacian(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<2> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<2> >(nstart, nx, nl,
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Trilaplacian(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (b1 == 0.0) {
    ierr = SpectralKernelCallGPU<NLaplacianKernel<3> >(nstart, nx, nl, 
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale); CHKERRQ(ierr);
  } else {
    ierr = SpectralKernelCallGPU<RelaxedNLaplacianKernel<3> >(nstart, nx, nl,
      pXHat[0], pXHat[1], pXHat[2], 
      b0*scale, b1); CHKERRQ(ierr);
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::TrilaplacianFunctional(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = ThrowError("trilaplacian operator not implemented"); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InverseLaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianSqrtKernel<1> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<1> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InverseBilaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      /// scale/sqrt(b0*|lapik|^2) = scale/(sqrt(b0)*|lapik|)
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<1> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, sqrt(b0)); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<2> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::InverseTrilaplacian(bool usesqrt, ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  if (usesqrt) {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianSqrtKernel<3> >(nstart, nx, nl,
        pXHat[0], pXHat[1], pXHat[2],
        scale, sqrt(b0)); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianSqrtKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  } else {
    if (b1 == 0.0) {
      ierr = SpectralKernelCallGPU<InverseNLaplacianKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2], 
        scale, b0); CHKERRQ(ierr);
    } else {
      ierr = SpectralKernelCallGPU<RelaxedInverseNLaplacianKernel<3> >(nstart, nx, nl, 
        pXHat[0], pXHat[1], pXHat[2],
        scale, b0, b1); CHKERRQ(ierr);
    }
  }

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Leray(ScalarType b0, ScalarType b1) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<LerayKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], 
    scale, b0, b1); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::GaussianFilter(const ScalarType c[3]) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<GaussianFilterKernel>(nstart, nx, nl, 
    pXHat[0], c[0], c[1], c[2], scale); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

PetscErrorCode DifferentiationKernel::Gradient() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<GradientKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], scale); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}
PetscErrorCode DifferentiationKernel::Divergence() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ierr = SpectralKernelCallGPU<DivergenceKernel>(nstart, nx, nl, 
    pXHat[0], pXHat[1], pXHat[2], scale); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

} // namespace reg

#endif
