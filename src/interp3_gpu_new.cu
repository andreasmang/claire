#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------*\
Copyright (c) 2008-2010, Danny Ruijters. All rights reserved.
http://www.dannyruijters.nl/cubicinterpolation/
This file is part of CUDA Cubic B-Spline Interpolation (CI).

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
*  Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
*  Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.
*  Neither the name of the copyright holders nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied.

When using this code in a scientific project, please cite one or all of the
following papers:
*  Daniel Ruijters and Philippe Thévenaz,
   GPU Prefilter for Accurate Cubic B-Spline Interpolation, 
   The Computer Journal, vol. 55, no. 1, pp. 15-20, January 2012.
   http://dannyruijters.nl/docs/cudaPrefilter3.pdf
*  Daniel Ruijters, Bart M. ter Haar Romeny, and Paul Suetens,
   Efficient GPU-Based Texture Interpolation using Uniform B-Splines,
   Journal of Graphics Tools, vol. 13, no. 4, pp. 61-69, 2008.
\*--------------------------------------------------------------------------*/

#include <stdio.h>
#include "petsc.h"
#include "petscconf.h"
#include "petsccuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <algorithm>
#include <thrust/device_ptr.h>

#include <memcpy.cu>
#include <cubicPrefilter3D.cu>
#include <bspline_kernel.cu>
#include <lagrange_kernel.cu>
#include "interp3_gpu_new.hpp"


#define PI ((double)3.14159265358979323846264338327950288419716939937510)
#define max(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
            _a > _b ? _a : _b; })

#define KERNEL_DIM 4
#define MAX_BLOCKS 1024

template <typename T>
__host__ __device__
inline T rec3_fmaf(T a, T b, T c, T d, T e, T f) {
    return fmaf(a, b, fmaf(c, d, e*f));
    //return a*b + (c*d + e*f);
}


template <typename T>
__host__ __device__
inline T rec4_fmaf(T a, T b, T c, T d, T e, T f, T g, T h) {
    return fmaf(a, b, fmaf(c, d, fmaf( e, f, g*h)));
}


/********************************************************************
 * @brief device function to do the interpolation of a single point using the Fast Lagrange Method
 * @parm[in] tex input data texture used for interpolation
 * @parm[in] coord_grid query coordinate
 * @parm[in] inv_reg_extent inverse of the dimension of the 3D grid (1/nx, 1/ny, 1/nz)
 * @parm[out] interpolated value
 *******************************************************************/
__device__ float cubicTex3D_lagrangeFast( hipTextureObject_t tex, const float3 coord_grid, const float3 inv_ext)
{
	const float3 index = floor(coord_grid);
	float3 w0, w1, w2, w3;
	lagrange_weights(coord_grid - index, w0, w1, w2, w3);
    
    // compute the locations for the trilinear, bilinear and linear interps
    const float3 g0 = w1 + w2;
    const float3 h0 = (w2/g0 + index + 0.5f)*inv_ext;
    float idx[2] = { (index.x-0.5f)*inv_ext.x, (index.x+2.5f)*inv_ext.x};
    float idy[2] = { (index.y-0.5f)*inv_ext.y, (index.y+2.5f)*inv_ext.y};
    float idz[2] = { (index.z-0.5f)*inv_ext.z, (index.z+2.5f)*inv_ext.z};

    // single trilinear lookup
    float core = tex3D<float>( tex, h0.x, h0.y, h0.z);

    // 6 bilinear lookups
    float z0 = tex3D<float>( tex, h0.x, h0.y, idz[0]);
    float z1 = tex3D<float>( tex, h0.x, h0.y, idz[1]);
    float y0 = tex3D<float>( tex, h0.x, idy[0], h0.z);
    float y1 = tex3D<float>( tex, h0.x, idy[1], h0.z);
    float x0 = tex3D<float>( tex, idx[0], h0.y, h0.z);
    float x1 = tex3D<float>( tex, idx[1], h0.y, h0.z);

    // 12 linear lookups
    // along z-axis
    float x0y0 = tex3D<float>( tex, idx[0], idy[0], h0.z);
    float x1y1 = tex3D<float>( tex, idx[1], idy[1], h0.z);
    float x0y1 = tex3D<float>( tex, idx[0], idy[1], h0.z);
    float x1y0 = tex3D<float>( tex, idx[1], idy[0], h0.z);
    // aling y-axis
    float x0z0 = tex3D<float>( tex, idx[0], h0.y, idz[0]);
    float x1z1 = tex3D<float>( tex, idx[1], h0.y, idz[1]);
    float x0z1 = tex3D<float>( tex, idx[0], h0.y, idz[1]);
    float x1z0 = tex3D<float>( tex, idx[1], h0.y, idz[0]);
    // along x-axis
    float y0z0 = tex3D<float>( tex, h0.x, idy[0], idz[0]);
    float y1z1 = tex3D<float>( tex, h0.x, idy[1], idz[1]);
    float y0z1 = tex3D<float>( tex, h0.x, idy[0], idz[1]);
    float y1z0 = tex3D<float>( tex, h0.x, idy[1], idz[0]);

    // 8 single point look ups
    float tex000 = tex3D<float>( tex, idx[0], idy[0], idz[0]);
    float tex100 = tex3D<float>( tex, idx[1], idy[0], idz[0]);
    float tex010 = tex3D<float>( tex, idx[0], idy[1], idz[0]);
    float tex110 = tex3D<float>( tex, idx[1], idy[1], idz[0]);
    float tex001 = tex3D<float>( tex, idx[0], idy[0], idz[1]);
    float tex101 = tex3D<float>( tex, idx[1], idy[0], idz[1]);
    float tex011 = tex3D<float>( tex, idx[0], idy[1], idz[1]);
    float tex111 = tex3D<float>( tex, idx[1], idy[1], idz[1]);

    // weighting in x direction
    // slice 1 (z=0)
    float row0 = rec3_fmaf( w0.x,  tex000,  g0.x,  y0z0,  w3.x,  tex100);
    float row1 = rec3_fmaf( w0.x,  x0z0,    g0.x,  z0,    w3.x,  x1z0);
    float row2 = rec3_fmaf( w0.x,  tex010,  g0.x,  y1z0,  w3.x,  tex110);
    // weighting along y direction
    float Z0 = rec3_fmaf( w0.y, row0, g0.y, row1, w3.y, row2);
    // slice 3 (z=1), weighing along x direction
    row0 = rec3_fmaf( w0.x, tex001, g0.x, y0z1, w3.x, tex101);
    row1 = rec3_fmaf( w0.x, x0z1,   g0.x, z1,   w3.x, x1z1);
    row2 = rec3_fmaf( w0.x, tex011, g0.x, y1z1, w3.x, tex111);
    // weighting along y direction
    float Z2 = rec3_fmaf( w0.y, row0, g0.y, row1, w3.y, row2);

    // slice 2 (z in middle, 4 bilinear, 4 linear and 1 trilinear lookup), weighing along x-direction
    row0 = rec3_fmaf( w0.x, x0y0, g0.x, y0, w3.x, x1y0);
    row1 = rec3_fmaf( w0.x, x0, g0.x, core, w3.x, x1);
    row2 = rec3_fmaf( w0.x, x1y0, g0.x, y1, w3.x, x1y1);
    // weighting along y direction
    float Z1 = rec3_fmaf( w0.y, row0, g0.y, row1, w3.y, row2);
    
    // weighting along z-direction
    return rec3_fmaf( w0.z, Z0, g0.z, Z1, w3.z, Z2);
}


/********************************************************************
 * @brief device function to do the interpolation of a single point using the Vanilla Lagrange Method
 * @parm[in] tex input data texture used for interpolation
 * @parm[in] coord_grid query coordinate
 * @parm[in] inv_reg_extent inverse of the dimension of the 3D grid (1/nx, 1/ny, 1/nz)
 * @parm[out] interpolated value
 *******************************************************************/
__device__ float cubicTex3D_lagrangeSimple(hipTextureObject_t tex, float3 coord, const float3 inv_ext)
{
	const float3 coord_grid = coord;
	const float3 index = floor(coord_grid);
	const float3 fraction = coord_grid - index;
	float3 w0, w1, w2, w3;
	lagrange_weights(fraction, w0, w1, w2, w3);

    float idx[KERNEL_DIM] = { (index.x-0.5f)*inv_ext.x, 
                     (index.x+0.5f)*inv_ext.x,
                     (index.x+1.5f)*inv_ext.x,
                     (index.x+2.5f)*inv_ext.x};
    
    float wx[KERNEL_DIM] = {w0.x, w1.x, w2.x, w3.x};

    float idy[KERNEL_DIM] = { (index.y-0.5f)*inv_ext.y, 
                     (index.y+0.5f)*inv_ext.y,
                     (index.y+1.5f)*inv_ext.y,
                     (index.y+2.5f)*inv_ext.y};
    float wy[KERNEL_DIM] = {w0.y, w1.y, w2.y, w3.y};

    float idz[KERNEL_DIM] = { (index.z-0.5f)*inv_ext.z, 
                     (index.z+0.5f)*inv_ext.z,
                     (index.z+1.5f)*inv_ext.z,
                     (index.z+2.5f)*inv_ext.z};
    float wz[KERNEL_DIM] = {w0.z, w1.z, w2.z, w3.z};
    
    float yq = 0;
    int j,k;
    float sj,sk;
    
    for(k=0; k<KERNEL_DIM; k++){
        sk = 0;
        for(j=0; j<KERNEL_DIM; j++){
            sj = rec4_fmaf( wx[0],  tex3D<float>(tex, idx[0], idy[j], idz[k]), 
                            wx[1],  tex3D<float>(tex, idx[1], idy[j], idz[k]),
                            wx[2],  tex3D<float>(tex, idx[2], idy[j], idz[k]),
                            wx[3],  tex3D<float>(tex, idx[3], idy[j], idz[k]));
    
            sk = fmaf(wy[j], sj, sk);
        }
        yq = fmaf(wz[k], sk, yq);
    }
    return yq;
}


/********************************************************************
 * @brief device function to do the interpolation of a single point using the Vanilla Spline Method
 * @parm[in] tex input data texture used for interpolation
 * @parm[in] coord_grid query coordinate
 * @parm[in] inv_reg_extent inverse of the dimension of the 3D grid (1/nx, 1/ny, 1/nz)
 * @parm[out] interpolated value
 *******************************************************************/
__device__ float cubicTex3D_splineSimple(hipTextureObject_t tex, float3 coord, const float3 inv_extent)
{
	// transform the coordinate from [0,extent] to [-0.5, extent-0.5]
	const float3 coord_grid = coord;
	float3 index = floor(coord_grid);
	const float3 fraction = coord_grid - index;
	index = index + 0.5f;  //move from [-0.5, extent-0.5] to [0, extent]

	float result = 0.0f;
	for (float z=-1; z < 2.5f; z++)  //range [-1, 2]
	{
		float bsplineZ = bspline(z-fraction.z);
		float w = (index.z + z)*inv_extent.z;
		for (float y=-1; y < 2.5f; y++)
		{
			float bsplineYZ = bspline(y-fraction.y) * bsplineZ;
			float v = (index.y + y)*inv_extent.y;
			for (float x=-1; x < 2.5f; x++)
			{
				float bsplineXYZ = bspline(x-fraction.x) * bsplineYZ;
				float u = (index.x + x)*inv_extent.z;
				result = fmaf(bsplineXYZ , tex3D<float>(tex, u, v, w), result);
			}
		}
	}
	return result;
}



/********************************************************************
 * @brief device function to do the interpolation of a single point using the Fast Spline Method
 * @parm[in] tex input data texture used for interpolation
 * @parm[in] coord_grid query coordinate
 * @parm[in] inv_reg_extent inverse of the dimension of the 3D grid (1/nx, 1/ny, 1/nz)
 * @parm[out] interpolated value
 *******************************************************************/
__device__ float cubicTex3D_splineFast(hipTextureObject_t tex, const float3 coord_grid, const float3 inv_reg_extent)
{
	// shift the coordinate from [0,extent] to [-0.5, extent-0.5]
	const float3 index = floor(coord_grid);
	const float3 fraction = coord_grid - index;
	float3 w0, w1, w2, w3;
	bspline_weights(fraction, w0, w1, w2, w3);

	const float3 g0 = w0 + w1;
	const float3 g1 = 1.0f - g0;
	const float3 h0 = ((w1 / g0) - 0.5f + index)*inv_reg_extent;
	const float3 h1 = ((w3 / g1) + 1.5f + index)*inv_reg_extent;
    

	// fetch the eight linear interpolations
	// weighting and fetching is interleaved for performance and stability reasons
	float tex000 = tex3D<float>(tex, h0.x, h0.y, h0.z);
	float tex100 = tex3D<float>(tex, h1.x, h0.y, h0.z);
	//tex000 = g0.x * tex000 + g1.x * tex100;  //weigh along the x-direction
	tex000 = lerp(tex100, tex000, g0.x);
	
	float tex010 = tex3D<float>(tex, h0.x, h1.y, h0.z);
	float tex110 = tex3D<float>(tex, h1.x, h1.y, h0.z);
	//tex010 = g0.x * tex010 + g1.x * tex110;  //weigh along the x-direction
	tex010 = lerp( tex110, tex010, g0.x);
	//tex000 = g0.y * tex000 + g1.y * tex010;  //weigh along the y-direction
	tex000 = lerp( tex010, tex000, g0.y);
	
	float tex001 = tex3D<float>(tex, h0.x, h0.y, h1.z);
	float tex101 = tex3D<float>(tex, h1.x, h0.y, h1.z);
	//tex001 = g0.x * tex001 + g1.x * tex101;  //weigh along the x-direction
	tex001 = lerp( tex101, tex001, g0.x);
	
	float tex011 = tex3D<float>(tex, h0.x, h1.y, h1.z);
	float tex111 = tex3D<float>(tex, h1.x, h1.y, h1.z);
	//tex011 = g0.x * tex011 + g1.x * tex111;  //weigh along the x-direction
	tex011 = lerp( tex111, tex011, g0.x);
	//tex001 = g0.y * tex001 + g1.y * tex011;  //weigh along the y-direction
    tex001 = lerp( tex011, tex001, g0.y);

	//return (g0.z * tex000 + g1.z * tex001);  //weigh along the z-direction
    return lerp( tex001, tex000, g0.z);
}



/********************************************************************
 * @brief function to create a 3D texture from the given cuda Pitched Pointer denoting volume (3D) data
 *******************************************************************/
extern "C" hipTextureObject_t initTextureFromVolume(hipPitchedPtr volume, hipExtent extent) {
   hipError_t err = hipSuccess;
   hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
   hipArray* cuArray;
   err = hipMalloc3DArray(&cuArray, &channelDesc, extent, 0);
   if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate 3D hipArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
   }
   
   hipMemcpy3DParms p = {0};
   p.srcPtr = volume;
   p.dstArray = cuArray;
   p.extent = extent;
   p.kind = hipMemcpyDeviceToDevice;
   err = hipMemcpy3D(&p);
   if (err != hipSuccess){
        fprintf(stderr, "Failed to copy 3D memory to hipArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
   }
    /* create cuda resource description */
    struct hipResourceDesc resDesc;
    memset( &resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.addressMode[2] = hipAddressModeWrap;
    texDesc.readMode = hipReadModeElementType;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    err = hipCreateTextureObject( &texObj, &resDesc, &texDesc, NULL);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to create texture (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return texObj;

}



/********************************************************************
 * @brief device function for computing the linear index from given 3D indices
 *******************************************************************/
__device__ int getLinearIdxfrom3DCoord(int x, int y, int z, int width, int height) {
    
    // width will be the pitch in case of pitched memory
    return  x +  width*y + width*height*z;
    
}

/********************************************************************
 * @brief device functions for computing the true velocity and function 
 * values at given coordinates for debugging purposes
 *******************************************************************/
__device__ float computeVx(float x, float y, float z) {
    return cosf(y)*cosf(z);
}

__device__ float computeVy(float x, float y, float z) {
    return sinf(x)*sinf(z);
}

__device__ float computeVz(float x, float y, float z) {
    return cosf(x)*cosf(y);
}

__device__ float computeM(float x, float y, float z) {
    return (sinf(x)*sinf(x) + sinf(y)*sinf(y) + sinf(z)*sinf(z))/3.0f;
}


/********************************************************************
 * @brief kernel function for computing the linear index from given 3D indices
 *******************************************************************/
__global__ void compareMemoryWithTexture( float* m, hipPitchedPtr mptr, hipTextureObject_t mtex, const float3 nx, const float3 inv_extent) {
    const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    const int tidz = blockIdx.z * blockDim.z + threadIdx.z;
    const int tid = tidx*nx.y*nx.z + tidy*nx.z + tidz;
    
    float3 q = make_float3(tidx+0.5, tidy+0.5, tidz+0.5);
    float3 qcoord = make_float3(tidx, tidy, tidz);
    qcoord *= 2*PI*inv_extent.x;
    q = q*inv_extent;
    float mval = m[tid];
    char* mptr1 = (char*)mptr.ptr;
    size_t pitch = mptr.pitch;
    size_t slicePitch = pitch*nx.y;
    char* slice = mptr1 + tidx*slicePitch;
    float* depth = (float*)(slice + tidy*pitch);
    float mptrval = depth[tidz]; 
    float mtexval = tex3D<float>(mtex, q.z, q.y, q.x);
    float mtrue = computeVx(qcoord.x, qcoord.y, qcoord.z);
    if (tid>=1 && tid<20) {
        printf("[%d,%d,%d] \t tid = %d \t  mval = %f \t mptrval = %f \t mtexval = %f\t mtrue = %f\n", tidx, tidy, tidz, tid, mval, mptrval, mtexval, mtrue);
    }
}


/********************************************************************
 * @brief interpolation kernel for scalar field
 * @parm[in] yi_tex 3D texture used for interpolation
 * @parm[in] xq,yq,zq query coordinates
 * @parm[in] nx array denoting number of query coordinates in each dimension 
 * @parm[out] yo memory for storing interpolated values
 *******************************************************************/
__global__ void interp3D_kernel(
        hipTextureObject_t  yi_tex,
        const PetscScalar* xq,
        const PetscScalar* yq,
        const PetscScalar* zq, 
        PetscScalar* yo,
        const float3 inv_nx)
{
    // Get thread index 
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float3 qcoord = make_float3(zq[tid], yq[tid], xq[tid]);
    // do single point interpolation - 4 methods

    yo[tid] = cubicTex3D_splineFast(yi_tex, qcoord, inv_nx);
    //yo[tid] = cubicTex3D_splineSimple(yi_tex, qcoord, inv_nx);
    //yo[tid] = cubicTex3D_lagrangeSimple(yi_tex, qcoord, inv_nx);
    //yo[tid] = cubicTex3D_lagrangeFast(yi_tex, qcoord, inv_nx);

/*    const float h = 2*PI*inv_nx.x;
    const float3 q = qcoord*h;
    float votrue = computeVx(q.z, q.y, q.x);
    if (tid>=60 && tid<70) {
        printf("tidz = %d  x = %f  y = %f  z = %f  vi = %f  vo = %f  votrue  = %f\n",tid, qcoord.x, qcoord.y, qcoord.z, *((float*)(yi.ptr)+tid), yo[tid], votrue);
    }
*/

}



/********************************************************************
 * @brief host function to do interpolation of a scalar field
 * @parm[in] yi input data values 
 * @parm[in] xq1,yq1,zq1 query coordinates
 * @parm[in] nx array denoting number of query coordinates in each dimension 
 * @parm[out] yo interpolated values
 * @parm[out] interp_time time for computing the interpolation
 *******************************************************************/
void gpuInterp3D(
           PetscScalar* yi,
           const PetscScalar* xq1,
           const PetscScalar* xq2,
           const PetscScalar* xq3,
           PetscScalar* yo,
           int*  nx,
           float* interp_time)
{
   
    // timing variables
    float time=0, dummy_time=0;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // define inv of nx for normalizing in texture interpolation
    const float3 inv_nx = make_float3(  1.0f/static_cast<float>(nx[2]),
                                        1.0f/static_cast<float>(nx[1]), 
                                        1.0f/static_cast<float>(nx[0]));
    // define nxq, the dimensions of the grid
    const float3 nxq = make_float3( nx[0], nx[1], nx[2]);
    long int nq = nx[0]*nx[1]*nx[2]; 

    // create a common hipResourceDesc objects
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
   
    // create cudaPitchedPointers for query points by copying them to another device location
    //hipPitchedPtr xq, yq, zq;
    //xq = CopyVolumeDeviceToDevice(xq1, nq, 1, 1);
    //yq = CopyVolumeDeviceToDevice(xq2, nq, 1, 1);
    //zq = CopyVolumeDeviceToDevice(xq3, nq, 1, 1);
    //////////////////////////////////////////////////////////////////////////////////////////////////    
    
   

    // make input image a hipPitchedPtr for fi
    hipPitchedPtr yi_cudaPitchedPtr = make_hipPitchedPtr(static_cast<void*>(yi), nx[2]*sizeof(float), nx[2], nx[1]);
    // initiate by computing the bspline coefficients for mt (in-place computation, updates mt)
    CubicBSplinePrefilter3D_Periodic((float*)yi_cudaPitchedPtr.ptr, (uint)yi_cudaPitchedPtr.pitch, nx[2], nx[1], nx[0]);
    // create a hipExtent for input resolution
    hipExtent yi_extent = make_hipExtent(nx[2], nx[1], nx[0]);
    // create a texture from the spline coefficients
    hipTextureObject_t yi_tex = initTextureFromVolume(yi_cudaPitchedPtr,  yi_extent);

    int threads = 256;
    int blocks = nq/threads;
    
    // check the correctness of the input data by checking the consistency across
    // the linear memory, pitched pointer and the texture
/*	dim3 dimBlock(1, 16, 16);
	dim3 dimGrid(nx[0] / dimBlock.x, nx[1] / dimBlock.y, nx[2] / dimBlock.z);
    compareMemoryWithTexture<<<dimBlock, dimGrid>>>(yi, yi_cudaPitchedPtr, yi_tex, nxq, inv_nx);
    if ( hipSuccess != hipGetLastError())
        printf("Error in running the interp3D kernel\n");
    hipDeviceSynchronize(); 
    printf("\n----------------------------------------------------------------------------------------------------------\n");
*/

    // start recording the interpolation kernel
    time = 0; dummy_time = 0; 
    hipEventRecord(startEvent,0); 
    
    // launch the interpolation kernel
    interp3D_kernel<<<blocks,threads>>>(yi_tex, xq1, xq2, xq3, yo, inv_nx);
    //interp3D_kernel<<<blocks,threads>>>(yi_tex, (PetscScalar*)xq.ptr, (PetscScalar*)yq.ptr, (PetscScalar*)zq.ptr, yo, inv_nx);
    if ( hipSuccess != hipGetLastError())
        printf("Error in running the interp3D kernel\n");

    hipEventRecord(stopEvent,0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dummy_time, startEvent, stopEvent);
    time+=dummy_time;
    hipDeviceSynchronize();
    
    // free texture and hipArray from device memory
    hipGetTextureObjectResourceDesc( &resDesc, yi_tex);
    hipDestroyTextureObject(yi_tex);
    hipFreeArray( resDesc.res.array.array);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    
    printf("interp time = %fmsec\n", time);
    *interp_time += time;
    
}


/********************************************************************
 * @brief kernel function to do get the initial condition i.e grid indices for SemiLagrangian 
 * @parm[out] x,y,z memory space for storing coordinates
 * @parm[in] nxq dimensions for the grid
 *******************************************************************/
__global__ void getSMLInitialCondition_kernel(PetscScalar* x, PetscScalar* y, PetscScalar* z, const int3 nx) {
     int tidx = blockIdx.x * blockDim.x + threadIdx.x;
     int tidy = blockIdx.y * blockDim.y + threadIdx.y;
     int tidz = blockIdx.z * blockDim.z + threadIdx.z;
     int tid = tidx*nx.y*nx.z + tidy*nx.z + tidz;

    x[tid] = static_cast<PetscScalar>(tidx);
    y[tid] = static_cast<PetscScalar>(tidy);
    z[tid] = static_cast<PetscScalar>(tidz);

    if (tid>=4094 && tid<4098) {
        printf("tid = %d \t x = %0.1f \t y = %0.1f \t z = %0.1f\n", tid, x[tid], y[tid], z[tid]);
    }
    
}


/********************************************************************
 * @brief host function to do get the initial condition i.e grid indices for SemiLagrangian 
 * @parm[out] x,y,z memory space for storing coordinates
 * @parm[out] compute_time time required to do the initialization
 *******************************************************************/
void getSemiLagrangianInitialCondition(PetscScalar* x, PetscScalar* y, PetscScalar* z, int* nx, PetscScalar* compute_time) {
    float time=0, dummy_time=0;
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    
    const int3 nxq = make_int3(nx[0], nx[1], nx[2]);

    dim3 dimBlock(1,16,16);
	dim3 dimGrid( nx[0] / dimBlock.x, nx[1] / dimBlock.y, nx[2] / dimBlock.z );
    
/*	uint dimX = min(min(PowTwoDivider(nx[0]), PowTwoDivider(nx[1])), 64);
	uint dimY = min(min(PowTwoDivider(nx[2]), PowTwoDivider(nx[1])), 512/dimX);
	dim3 dimBlock(dimX, dimY);
    
	// Replace the voxel values by the b-spline coefficients
	dim3 dimGrid(nx[1] / dimBlock.x, nx[2] / dimBlock.y);
*/    
    printf("blockx = %d, blocky = %d, gridx - %d, gridy = %d\n", dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y);
  getSMLInitialCondition_kernel<<<dimBlock, dimGrid>>>(x, y, z, nxq);
    
    if ( hipSuccess != hipGetLastError())
        printf("Error in running the SML initial condition kernel\n");
    
    hipEventRecord(stopEvent,0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&dummy_time, startEvent, stopEvent);
    time+=dummy_time/1000;
    hipDeviceSynchronize();

    *compute_time += time;
}
