#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2018.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE. If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#include "DistanceMeasureKernel.hpp"
#include "cuda_helper.hpp"
#include "thrust/device_ptr.h"
#include "thrust/reduce.h"
#include "thrust/execution_policy.h"


__global__ void VecSubMulGPU(ScalarType *pL, const ScalarType *pW, const ScalarType *pWts,
    const ScalarType *pMr, const ScalarType *pM, int nl) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int k = blockIdx.y*nl + i;
  
  if (i < nl) {
    pL[k] = pW[i] * pWts[blockIdx.y] * (pMr[k] - pM[k]);
  }
}

__global__ void VecSubGPU(ScalarType *pL, const ScalarType *pWts, const ScalarType *pMr, 
    const ScalarType *pM, int nl) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int k = blockIdx.y*nl + i;
  
  if (i < nl) {
    pL[k] = pWts[blockIdx.y] * (pMr[k] - pM[k]);
  }
}

__global__ void VecMulGPU(ScalarType *pL, const ScalarType *pW, const ScalarType *pWts,
    const ScalarType *pM, int nl) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int k = blockIdx.y*nl + i;
  
  if (i < nl) {
    pL[k] = - pW[i] * pWts[blockIdx.y] * pM[k];
  }
}

__global__ void VecNegGPU(ScalarType *pL, const ScalarType *pWts, const ScalarType *pM, int nl) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int k = blockIdx.y*nl + i;
  
  if (i < nl) {
    pL[k] = - pWts[blockIdx.y] * pM[k];
  }
}

// TODO I would rather use cuBlas functions for reduction for performace reasons
template<int N> inline __device__ void LocalReductionSum(ScalarType *shared) {
  if (threadIdx.x < N) {
    shared[threadIdx.x] += shared[threadIdx.x + N];
  }
  __syncthreads();
  LocalReductionSum<N/2>(shared);
}

template<> inline __device__ void LocalReductionSum<1>(ScalarType *shared) {
  if (threadIdx.x == 0) {
    shared[0] += shared[1];
  }
  __syncthreads();
}

template<int N> __global__ void ReductionSum(ScalarType *res, int n) {
  __shared__ ScalarType value[N];
  
  value[threadIdx.x] = 0.0;
  for (int i=threadIdx.x; i<n; i+=N) {
    value[threadIdx.x] += res[i];
  }
  
  __syncthreads();
  
  LocalReductionSum<N/2>(value);
  
  if (threadIdx.x == 0) {
    res[0] = value[0];
  }
}

template<int N>
__global__ void DistanceMeasureFunctionalGPU(ScalarType *res, 
    const ScalarType *pW, const ScalarType *pWts, 
    const ScalarType *pMr,const ScalarType *pM, 
    int nl) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int k = blockIdx.y*nl + i;
  
  __shared__ ScalarType value[N];
  
  ScalarType tmp;
  
  value[threadIdx.x] = 0.0;
  
  if (i < nl) {
    tmp = pMr[k] - pM[k];
    value[threadIdx.x] = tmp*tmp*pW[i]*pWts[blockIdx.y];
  }
  
  __syncthreads();
    
  LocalReductionSum<N/2>(value);
  
  if (threadIdx.x == 0) {
    res[blockIdx.x + blockIdx.y*gridDim.x] = value[0];
  }
}

template<int N>
__global__ void DistanceMeasureFunctionalGPU(ScalarType *res, 
    const ScalarType *pWts, const ScalarType *pMr, const ScalarType *pM, int nl) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int k = blockIdx.y*nl + i;
  
  __shared__ ScalarType value[N];
  ScalarType tmp;
  
  value[threadIdx.x] = 0.0;  
  if (i < nl) {
    tmp = (pMr[k] - pM[k]);
    value[threadIdx.x] = tmp*tmp*pWts[blockIdx.y];
  }
  
  __syncthreads();
    
  LocalReductionSum<N/2>(value);
  
  if (threadIdx.x == 0) {
    res[blockIdx.x + blockIdx.y*gridDim.x] = value[0];
  }
}

namespace reg {
namespace DistanceMeasureKernel {

/* Compute Masked Registration Functional */
PetscErrorCode EvaluateFunctionalSL2::ComputeFunctionalMask() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  ScalarType *res = nullptr;
  PetscFunctionBegin;
  
  ierr = AllocateMemoryOnce(res, grid.x*grid.y*sizeof(ScalarType)); CHKERRQ(ierr);
  
  DistanceMeasureFunctionalGPU<256><<<grid, block>>>(res, pW, pWts, pMr, pM, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  ReductionSum<1024><<<1, 1024>>>(res, grid.x*grid.y);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  ierr = hipMemcpy(reinterpret_cast<void*>(&value), reinterpret_cast<void*>(res), sizeof(ScalarType), hipMemcpyDeviceToHost); CHKERRCUDA(ierr);
    
  FreeMemory(res);
  
  PetscFunctionReturn(ierr);
}

/* Compute the Registration Functional */
PetscErrorCode EvaluateFunctionalSL2::ComputeFunctional() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  ScalarType *res = nullptr;
  PetscFunctionBegin;
  
  ierr = AllocateMemoryOnce(res, grid.x*grid.y*sizeof(ScalarType)); CHKERRQ(ierr);
  
  DistanceMeasureFunctionalGPU<256><<<grid, block>>>(res, pWts, pMr, pM, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  ReductionSum<1024><<<1, 1024>>>(res, grid.x*grid.y);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  ierr = hipMemcpy(reinterpret_cast<void*>(&value), reinterpret_cast<void*>(res), sizeof(ScalarType), hipMemcpyDeviceToHost); CHKERRCUDA(ierr);
    
  FreeMemory(res);
  
  PetscFunctionReturn(ierr);
}

/* Final Condition for Adjoint Equation */
PetscErrorCode FinalConditionSL2::ComputeFinalConditionAE() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  PetscFunctionBegin;
  
  VecSubGPU<<<grid, block>>>(pL, pWts, pMr, pM, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  PetscFunctionReturn(ierr);
}

/* Final Condition for Masked Adjoint Equation */
PetscErrorCode FinalConditionSL2::ComputeFinalConditionMaskAE() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  PetscFunctionBegin;
  
  VecSubMulGPU<<<grid, block>>>(pL, pW, pWts, pMr, pM, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  PetscFunctionReturn(ierr);
}

/* Final Condition for Incremental Adjoint Equation */
PetscErrorCode FinalConditionSL2::ComputeFinalConditionIAE() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  PetscFunctionBegin;
  
  VecNegGPU<<<grid, block>>>(pL, pWts, pM, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  hipblasStatus_t stat;
  hipblasHandle_t handle; 
  stat = hipblasCreate(&handle);
  stat = hipblasSnrm2(handle, nl*nc, pM, 1, &norm_mtilde_loc);
  stat = hipblasDestroy(handle);

  
  PetscFunctionReturn(ierr);
}

/* Final Condition for Masked Incremental Adjoint Equation */
PetscErrorCode FinalConditionSL2::ComputeFinalConditionMaskIAE() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  PetscFunctionBegin;
 
  VecMulGPU<<<grid, block>>>(pL, pW, pWts, pM, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  PetscFunctionReturn(ierr);
}


////////////////////////////////////////////////////////////////////////
//> NCC Distance metric routines 
///////////////////////////////////////////////////////////////////////
__global__ void FinalConditionAENCC_kernel (ScalarType *pL, const ScalarType *pMr, const ScalarType *pM, ScalarType const1, ScalarType const2, ScalarType const3) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  pL[i] = const1*pMr[i] - const2*pM[i] + const3;
}

__global__ void FinalConditionIAENCC_kernel (ScalarType *pLtilde, const ScalarType *pMr, const ScalarType *pM, const ScalarType *pMtilde, ScalarType const1tilde, ScalarType const3tilde, ScalarType const5, ScalarType mean_m1, ScalarType mean_mR) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  pLtilde[i] = const1tilde*(pMr[i]-mean_mR) + const3tilde*(pM[i]-mean_m1) - const5*pMtilde[i];
}

/* Compute the Registration Functional */
PetscErrorCode EvaluateFunctionalNCC::ComputeScaleMask() {
  PetscErrorCode ierr = 0;
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  ScalarType *res = nullptr;
  PetscFunctionBegin;
  
  // not implemented
  
  PetscFunctionReturn(ierr);
}

/* Compute the Registration Functional */
PetscErrorCode EvaluateFunctionalNCC::ComputeScale() {
  PetscErrorCode ierr = 0;
  hipblasStatus_t stat;
  hipblasHandle_t handle; 
  dim3 block(256, 1, 1);
  dim3 grid((nl + 255)/256, nc, 1);
  ScalarType *res = nullptr;
  ScalarType sum = 0.0;
  PetscFunctionBegin;
  
  // compute local sums
  sum_mT_loc = thrust::reduce(thrust::device, pMt, pMt+nl*nc);
  sum_mR_loc = thrust::reduce(thrust::device, pMr, pMr+nl*nc);
  
  stat = hipblasCreate(&handle);

  stat = hipblasSnrm2(handle, nl*nc, pMt, 1, &norm_mT_loc);
  norm_mT_loc *= norm_mT_loc;
  stat = hipblasSnrm2(handle, nl*nc, pMr, 1, &norm_mR_loc);
  norm_mR_loc *= norm_mR_loc;
  stat = hipblasSdot(handle, nl*nc, pMt, 1, pMr, 1, &inpr_mT_mR_loc);

  stat = hipblasDestroy(handle);
  
  ierr = AllocateMemoryOnce(res, grid.x*grid.y*sizeof(ScalarType)); CHKERRQ(ierr);
  
  DistanceMeasureFunctionalGPU<256><<<grid, block>>>(res, pWts, pMr, pMt, nl);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  ReductionSum<1024><<<1, 1024>>>(res, grid.x*grid.y);
  hipDeviceSynchronize();
  cudaCheckKernelError();
  
  ierr = hipMemcpy(reinterpret_cast<void*>(&norm_l2_loc), reinterpret_cast<void*>(res), sizeof(ScalarType), hipMemcpyDeviceToHost); CHKERRCUDA(ierr);
    
  FreeMemory(res);
  
  PetscFunctionReturn(ierr);
}

/* Compute the Registration Functional */
PetscErrorCode EvaluateFunctionalNCC::ComputeFunctional() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  // compute local sums
  sum_mR_loc = thrust::reduce(thrust::device, pMr, pMr+nl*nc);
  sum_m1_loc = thrust::reduce(thrust::device, pM, pM+nl*nc);

  hipblasStatus_t stat;
  hipblasHandle_t handle; 

  stat = hipblasCreate(&handle);

  stat = hipblasSnrm2(handle, nl*nc, pM, 1, &norm_m1_loc);
  norm_m1_loc *= norm_m1_loc;
  stat = hipblasSnrm2(handle, nl*nc, pMr, 1, &norm_mR_loc);
  norm_mR_loc *= norm_mR_loc;
  stat = hipblasSdot(handle, nl*nc, pM, 1, pMr, 1, &inpr_m1_mR_loc);

  stat = hipblasDestroy(handle);
  
  PetscFunctionReturn(ierr);
}

/* Compute the Registration Functional */
PetscErrorCode EvaluateFunctionalNCC::ComputeFunctionalMask() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
    
  // not implemented 
  PetscFunctionReturn(ierr);
}

/* Final Condition for Adjoint Equation */
PetscErrorCode FinalConditionNCC::ComputeFinalConditionAE() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  dim3 block(256,1,1);
  dim3 grid((nl*nc + 255)/256,1,1);

  FinalConditionAENCC_kernel<<<grid, block>>>(pL, pMr, pM, const1, const2, const3);
  cudaCheckKernelError();
  
  PetscFunctionReturn(ierr);
}

/* Final Condition for Adjoint Equation */
PetscErrorCode FinalConditionNCC::ComputeFinalConditionMaskAE() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  // not implemented 
  PetscFunctionReturn(ierr);
}

PetscErrorCode FinalConditionNCC::ComputeInnerProductsFinalConditionAE() {
  PetscErrorCode ierr = 0;
  hipblasStatus_t stat;
  hipblasHandle_t handle; 
  PetscFunctionBegin;

  // compute local sums
  sum_mR_loc = thrust::reduce(thrust::device, pMr, pMr+nl*nc);
  sum_m1_loc = thrust::reduce(thrust::device, pM, pM+nl*nc);
  
  stat = hipblasCreate(&handle);
  
  stat = hipblasSdot(handle, nl*nc, pM, 1, pM, 1, &norm_m1_loc);
  stat = hipblasSdot(handle, nl*nc, pMr, 1, pMr, 1, &norm_mR_loc);
  stat = hipblasSdot(handle, nl*nc, pM, 1, pMr, 1, &inpr_m1_mR_loc);
  
  stat = hipblasDestroy(handle);

  PetscFunctionReturn(ierr);
}

PetscErrorCode FinalConditionNCC::ComputeInnerProductsFinalConditionIAE() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  sum_mR_loc = thrust::reduce(thrust::device, pMr, pMr+nl*nc);
  sum_m1_loc = thrust::reduce(thrust::device, pM, pM+nl*nc);
  sum_mtilde_loc = thrust::reduce(thrust::device, pMtilde, pMtilde+nl*nc);

  hipblasStatus_t stat;
  hipblasHandle_t handle; 
  stat = hipblasCreate(&handle);

  ScalarType norm_mtilde_loc = 0;
  stat = hipblasSnrm2(handle, nl*nc, pMtilde, 1, &norm_mtilde_loc);
  norm_mtilde_loc *= norm_mtilde_loc;
  
  stat = hipblasSdot(handle, nl*nc, pMr, 1, pMtilde, 1, &inpr_mR_mtilde_loc);
  ierr = Assert(!PetscIsNanReal(inpr_mR_mtilde_loc), "is nan"); CHKERRQ(ierr);
  stat = hipblasSdot(handle, nl*nc, pM, 1, pMtilde, 1, &inpr_m1_mtilde_loc);
  ierr = Assert(!PetscIsNanReal(inpr_m1_mtilde_loc), "is nan"); CHKERRQ(ierr);
  stat = hipblasSnrm2(handle, nl*nc, pM, 1, &norm_m1_loc);
  norm_m1_loc *= norm_m1_loc;
  ierr = Assert(!PetscIsNanReal(norm_m1_loc), "is nan"); CHKERRQ(ierr);
  stat = hipblasSnrm2(handle, nl*nc, pMr, 1, &norm_mR_loc);
  norm_mR_loc *= norm_mR_loc;
  ierr = Assert(!PetscIsNanReal(norm_mR_loc), "is nan"); CHKERRQ(ierr);
  stat = hipblasSdot(handle, nl*nc, pM, 1, pMr, 1, &inpr_m1_mR_loc);
  ierr = Assert(!PetscIsNanReal(inpr_m1_mR_loc), "is nan"); CHKERRQ(ierr);
  
  stat = hipblasDestroy(handle);

  PetscFunctionReturn(ierr);
}

/* Final Condition for Incremental Adjoint Equation */
PetscErrorCode FinalConditionNCC::ComputeFinalConditionIAE(ScalarType mean_m1, ScalarType mean_mR) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  dim3 block(256,1,1);
  dim3 grid((nl*nc + 255)/256,1,1);
    
  ScalarType const1tilde = const1 - const2;
  ScalarType const3tilde = const3 - const4;
  FinalConditionIAENCC_kernel<<<grid, block>>>(pLtilde, pMr, pM, pMtilde, const1tilde, const3tilde, const5, mean_m1, mean_mR);
  cudaCheckKernelError();
  
  PetscFunctionReturn(ierr);
}

/* Final Condition for Incremental Adjoint Equation */
PetscErrorCode FinalConditionNCC::ComputeFinalConditionMaskIAE() {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  // not implemented
  
  PetscFunctionReturn(ierr);
}

} // namespace DistanceMeasureKernel
} // namespace reg
