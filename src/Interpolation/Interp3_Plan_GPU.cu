#include "hip/hip_runtime.h"
#include <interp3_gpu_mpi.hpp>
#include <string.h>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <cuda_helper.hpp>
#include <cmath>
#include <time.h>

#include <algorithm>

static void printGPUMemory(int rank) {
    if (rank == 0) {
      size_t free, used;
      hipMemGetInfo(&free, &used);
      used -= free;
      std::string msg = "Used mem = " + std::to_string(used/1E9) + " GB, Free mem = " + std::to_string(free/1E9) + " GB\n";
      PetscPrintf(PETSC_COMM_WORLD, msg.c_str());
    }
}
//#define VERBOSE1

#ifndef ACCFFT_CHECKCUDA_H
#define ACCFFT_CHECKCUDA_H
inline hipError_t checkCuda_accfft(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}
inline hipfftResult checkCuda_accfft(hipfftResult result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", result);
    assert(result == HIPFFT_SUCCESS);
  }
#endif
  return result;
}
#endif

inline size_t get_max_query_allocation(int* isize, int neighbour_query_width) {
  return (isize[0]+2*neighbour_query_width)*isize[1]*isize[2]; 
}

struct is_equal {
    int id;
    is_equal(int comp_id) : id(comp_id) {};
    __host__ __device__ 
    bool operator()(const int &x) {
        return x == id;
    }
};

template <typename Iterator>
class strided_range
{
    public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type,difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
        difference_type operator()(const difference_type& i) const
        { 
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator,TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}
   
    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }
    
    protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

class Trip_GPU{
  public:
    Trip_GPU(){};
    double x;
    double y;
    double z;
    int ind;
    int N[3];
    double h[3];

};

static bool ValueCmp(Trip_GPU const & a, Trip_GPU const & b)
{
    return a.z + a.y/a.h[1]*a.N[2] + a.x/a.h[0]* a.N[1]*a.N[2]<b.z + b.y/b.h[1]*b.N[2] + b.x/b.h[0]* b.N[1]*b.N[2] ;
}

#ifdef SORT_QUERIES
static void sort_queries(std::vector<Real>* query_outside,std::vector<int>* f_index,int* N_reg,Real* h,MPI_Comm c_comm){

  int nprocs, procid;
  MPI_Comm_rank(c_comm, &procid);
  MPI_Comm_size(c_comm, &nprocs);
  for(int proc=0;proc<nprocs;++proc){
    int qsize=query_outside[proc].size()/COORD_DIM;
    Trip_GPU* trip=new Trip_GPU[qsize];

    for(int i=0;i<qsize;++i){
      trip[i].x=query_outside[proc][i*COORD_DIM+0];
      trip[i].y=query_outside[proc][i*COORD_DIM+1];
      trip[i].z=query_outside[proc][i*COORD_DIM+2];
      trip[i].ind=f_index[proc][i];
      trip[i].N[0]=N_reg[0];
      trip[i].N[1]=N_reg[1];
      trip[i].N[2]=N_reg[2];
      trip[i].h[0]=h[0];
      trip[i].h[1]=h[1];
      trip[i].h[2]=h[2];
    }

    std::sort(trip, trip + qsize, ValueCmp);

    query_outside[proc].clear();
    f_index[proc].clear();

    for(int i=0;i<qsize;++i){
      query_outside[proc].push_back(trip[i].x);
      query_outside[proc].push_back(trip[i].y);
      query_outside[proc].push_back(trip[i].z);
      f_index[proc].push_back(trip[i].ind);
    }
    delete[] trip;
  }
  return;
}
#endif


Interp3_Plan_GPU::Interp3_Plan_GPU (size_t g_alloc_max) {
  this->g_alloc_max=g_alloc_max;
  this->allocate_baked=false;
  this->scatter_baked=false;
}


void Interp3_Plan_GPU::allocate (int N_pts, int data_dof)
{
  int nprocs, procid;
  MPI_Comm_rank(MPI_COMM_WORLD, &procid);
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
  query_points=(Real*) malloc(N_pts*COORD_DIM*sizeof(Real));

  f_index_procs_others_offset=(int*)malloc(nprocs*sizeof(int)); // offset in the all_query_points array
  f_index_procs_self_offset  =(int*)malloc(nprocs*sizeof(int)); // offset in the query_outside array
  f_index_procs_self_sizes   =(int*)malloc(nprocs*sizeof(int)); // sizes of the number of interpolations that need to be sent to procs
  f_index_procs_others_sizes =(int*)malloc(nprocs*sizeof(int)); // sizes of the number of interpolations that need to be received from procs

  s_request= new MPI_Request[nprocs];
  request= new MPI_Request[nprocs];
    
  //f_index = new thrust::device_vector<int> [nprocs];
  //query_outside=new thrust::device_vector<Real> [nprocs];

  query_outside = thrust::device_malloc<Real>(COORD_DIM*N_pts);
  f_index = thrust::device_malloc<int>(N_pts);
  num_query_per_proc.resize(nprocs);
  query_outside_offset.resize(nprocs);
  f_index_offset.resize(nprocs);

  //thrust::device_vector<int> f_index(N_pts);
  //thrust::device_vector<Real> query_outside(COORD_DIM*N_pts);
    
  // on CPU, N_pts = nl (number of local points), data_dof = 1 (Scalar field) / 3 (vector field)
  // The reshuffled semi-final interpolated values are stored here
  //f_cubic_unordered=(Real*) malloc(N_pts*sizeof(Real)*data_dof); 
  hipMalloc((void**)&f_cubic_unordered_d, N_pts*sizeof(Real)*data_dof);

  hipMalloc((void**)&query_points_x, sizeof(Real)*N_pts);
  hipMalloc((void**)&query_points_y, sizeof(Real)*N_pts);
  hipMalloc((void**)&query_points_z, sizeof(Real)*N_pts);
  hipMalloc((void**)&which_proc, sizeof(int)*N_pts);
    
  //double time=0;
  //time=-MPI_Wtime();

// Allocate memory for the ghost padded regular grid values
//#ifdef INTERP_PINNED
//  //hipHostMalloc((void**)&this->ghost_reg_grid_vals_d,g_alloc_max*data_dof);
//  hipMalloc((void**)&this->ghost_reg_grid_vals_d, g_alloc_max*data_dof);
//#else
//  hipMalloc((void**)&this->ghost_reg_grid_vals_d, g_alloc_max*data_dof);
//#endif

  //time+=MPI_Wtime();
  //if(procid==0)
  //  std::cout<<"malloc time="<<time<<std::endl;

  stype= new MPI_Datatype[nprocs];
  rtype= new MPI_Datatype[nprocs];
  this->data_dof=data_dof;
  this->allocate_baked=true;
}

Interp3_Plan_GPU::~Interp3_Plan_GPU ()
{
  int nprocs, procid;
  MPI_Comm_rank(MPI_COMM_WORLD, &procid);
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

  if(this->allocate_baked){
   // free(query_points);

    free(f_index_procs_others_offset);
    free(f_index_procs_self_offset  );
    free(f_index_procs_self_sizes   );
    free(f_index_procs_others_sizes );

    delete(s_request);
    delete(request);
    //vectors
    //for(int proc=0;proc<nprocs;++proc)
    //{
    //  thrust::device_vector<int>().swap(f_index[proc]);
    //  thrust::device_vector<Real>().swap(query_outside[proc]);
    //}
    hipFree(f_cubic_unordered_d);

    thrust::device_free(f_index);
    thrust::device_free(query_outside);
  }

  if(this->scatter_baked) {
    //free(all_query_points);

#ifdef INTERP_PINNED
    hipHostFree(all_f_cubic_d);
    hipHostFree(xq1);
    hipHostFree(xq2);
    hipHostFree(xq3);
    hipHostFree(all_query_points_d);
#else
    hipFree(all_f_cubic_d);
    hipFree(xq1);
    hipFree(xq2);
    hipFree(xq3);
    hipFree(all_query_points_d);
#endif

    for(int i=0;i<nprocs;++i) {
      MPI_Type_free(&stype[i]);
      MPI_Type_free(&rtype[i]);
    }

  }
  
    hipFree(query_points_x);
    hipFree(query_points_y);
    hipFree(query_points_z);
    hipFree(which_proc);

  if(this->allocate_baked) {
    delete(stype);
    delete(rtype);
  }
  return;
}

void rescale_xyz(const int g_size,  int* N_reg, int* N_reg_g, int* istart, const int N_pts, Real* query_points);


/*
 * Phase 1 of the parallel interpolation: This function computes which query_points needs to be sent to
 * other processors and which ones can be interpolated locally. Then a sparse alltoall is performed and
 * all the necessary information is sent/received including the coordinates for the query_points.
 * At the end, each process has the coordinates for interpolation of its own data and those of the others.
 *
 * IMPORTANT: This function must be called just once for a specific query_points. The reason is because of the
 * optimizations performed which assumes that the query_points do not change. For repeated interpolation you should
 * just call this function once, and instead repeatedly call Interp3_Plan::interpolate function.
 */
void Interp3_Plan_GPU::scatter( int data_dof,
                                int* N_reg,  // global grid dimensions
                                int * isize, // local grid dimensions
                                int* istart, // local grid start indices
                                const int N_pts, // local grid point count
                                const int g_size, // ghost layer width
                                Real* query_points_in_x, // input query points
                                Real* query_points_in_y, // input query points
                                Real* query_points_in_z, // input query points
                                int* c_dims,  // process cartesian grid dimensions
                                MPI_Comm c_comm,  // MPI Comm
                                double * timings) 
{
  int nprocs, procid;
  MPI_Comm_rank(c_comm, &procid);
  MPI_Comm_size(c_comm, &nprocs);

  if(this->allocate_baked==false){
    std::cout<<"ERROR Interp3_Plan_GPU Scatter called before calling allocate.\n";
    return;
  }
  //if(this->scatter_baked==true) {
  //  for(int proc=0;proc<nprocs;++proc) {
  //    thrust::device_vector<int>().swap(f_index[proc]);
  //    thrust::device_vector<Real>().swap(query_outside[proc]);
  //  }
  //}
  all_query_points_allocation=0;
 {
    //int N_reg_g[3], isize_g[3];
    N_reg_g[0]=N_reg[0]+2*g_size;
    N_reg_g[1]=N_reg[1]+2*g_size;
    N_reg_g[2]=N_reg[2]+2*g_size;

    isize_g[0]=isize[0]+2*g_size;
    isize_g[1]=isize[1]+2*g_size;
    isize_g[2]=isize[2]+2*g_size;

    Real h[3]; // original grid size along each axis
    h[0]=1./N_reg[0];
    h[1]=1./N_reg[1];
    h[2]=1./N_reg[2];
    
    //Real *query_points_x, *query_points_y, *query_points_z;
    //hipMalloc((void**)&query_points_x, sizeof(Real)*N_pts);
    //hipMalloc((void**)&query_points_y, sizeof(Real)*N_pts);
    //hipMalloc((void**)&query_points_z, sizeof(Real)*N_pts);
    ZeitGeist_define(scatter_memcpy);
    ZeitGeist_tick(scatter_memcpy);
    hipMemcpy(query_points_x, query_points_in_x, sizeof(Real)*N_pts, hipMemcpyDeviceToDevice);
    hipMemcpy(query_points_y, query_points_in_y, sizeof(Real)*N_pts, hipMemcpyDeviceToDevice);
    hipMemcpy(query_points_z, query_points_in_z, sizeof(Real)*N_pts, hipMemcpyDeviceToDevice);
    ZeitGeist_tock(scatter_memcpy);


    ZeitGeist_define(scatter_create_mpi_buffer);
    ZeitGeist_tick(scatter_create_mpi_buffer);
    // Enforce periodicity // write kernel for this
    timings[3]+=-MPI_Wtime();
    enforcePeriodicity(query_points_x, query_points_y, query_points_z, h, N_pts);
    timings[3]+=+MPI_Wtime();
    ZeitGeist_tock(scatter_create_mpi_buffer);
    
    //for(int i=0;i<N_pts;i++){
    //  while(query_points[i*COORD_DIM+0]<=-h[0]) {query_points[i*COORD_DIM+0]=query_points[i*COORD_DIM+0]+1;}
    //  while(query_points[i*COORD_DIM+1]<=-h[1]) {query_points[i*COORD_DIM+1]=query_points[i*COORD_DIM+1]+1;}
    //  while(query_points[i*COORD_DIM+2]<=-h[2]) {query_points[i*COORD_DIM+2]=query_points[i*COORD_DIM+2]+1;}

    //  while(query_points[i*COORD_DIM+0]>=1) {query_points[i*COORD_DIM+0]=query_points[i*COORD_DIM+0]-1;}
    //  while(query_points[i*COORD_DIM+1]>=1) {query_points[i*COORD_DIM+1]=query_points[i*COORD_DIM+1]-1;}
    //  while(query_points[i*COORD_DIM+2]>=1) {query_points[i*COORD_DIM+2]=query_points[i*COORD_DIM+2]-1;}
    //} 
    thrust::device_ptr<ScalarType> query_points_x_ptr = thrust::device_pointer_cast<ScalarType>(query_points_x);
    thrust::device_ptr<ScalarType> query_points_y_ptr = thrust::device_pointer_cast<ScalarType>(query_points_y);
    thrust::device_ptr<ScalarType> query_points_z_ptr = thrust::device_pointer_cast<ScalarType>(query_points_z);

    // Compute the start and end coordinates that this processor owns
    Real iX0[3],iX1[3];
    for (int j=0;j<3;j++) {
      iX0[j]=istart[j]*h[j];
      iX1[j]=iX0[j]+(isize[j]-1)*h[j];
    }

    // Now march through the query points and split them into nprocs parts.
    // These are stored in query_outside which is an array of vectors of size nprocs.
    // That is query_outside[i] is a vector that contains the query points that need to
    // be sent to process i. Obviously for the case of query_outside[procid], we do not
    // need to send it to any other processor, as we own the necessary information locally,
    // and interpolation can be done locally.


    // This is needed for one-to-one correspondence with output f. This is becaues we are reshuffling
    // the data according to which processor it land onto, and we need to somehow keep the original
    // index to write the interpolation data back to the right location in the output.

    // This is necessary because when we want to compute dproc0 and dproc1 we have to divide by
    // the max isize. If the proc grid is unbalanced, the last proc's isize will be different
    // than others. With this approach we always use the right isize0 for all procs.
    int isize0=std::ceil(N_reg[0]*1./c_dims[0]);
    int isize1=std::ceil(N_reg[1]*1./c_dims[1]);
    
    // number of coordinates to be sent to each proc
    int coords_in_proc;
    typedef thrust::device_vector<ScalarType>::iterator Iterator;
    
    ZeitGeist_tick(scatter_create_mpi_buffer);
    timings[3]+=-MPI_Wtime();
    checkDomain(which_proc, query_points_x, query_points_y, query_points_z, iX0, iX1, h, N_pts, procid, isize0, isize1, c_dims[1]);
    ZeitGeist_tock(scatter_create_mpi_buffer);

    thrust::device_ptr<int> which_proc_ptr = thrust::device_pointer_cast<int>(which_proc);
    
    ZeitGeist_define(scatter_memalloc);
    
    // loop over all procs
    f_index_offset[0] = 0;
    query_outside_offset[0] = 0;
    for (int proc=0; proc<nprocs; ++proc) {
        // count how many points belong to proc, will be useful in memory allocation
        ZeitGeist_tick(scatter_create_mpi_buffer);
        get_count(which_proc, N_pts, proc, &coords_in_proc);
        ZeitGeist_tock(scatter_create_mpi_buffer);

#if defined(VERBOSE1) 
        if (procid==0)
        PetscPrintf(PETSC_COMM_WORLD, "proc 0 sending %d points to proc %d\n", coords_in_proc, proc);
#endif
        
        num_query_per_proc[proc] = coords_in_proc;
        if (proc < nprocs-1) {
          f_index_offset[proc+1] = f_index_offset[proc] + coords_in_proc;
          query_outside_offset[proc+1] = query_outside_offset[proc] + coords_in_proc*COORD_DIM;
        }


        if (coords_in_proc > 0) {
            // get indices of coordinates which belong to this proc and store in f_index[proc]
            ZeitGeist_tick(scatter_create_mpi_buffer);
            //thrust::copy_if(thrust::device, thrust::make_counting_iterator(0), thrust::make_counting_iterator(N_pts), which_proc_ptr, f_index[proc].begin(), is_equal(proc));
            thrust::copy_if(thrust::device, thrust::make_counting_iterator(0), thrust::make_counting_iterator(N_pts), which_proc_ptr, f_index+f_index_offset[proc], is_equal(proc));
            ZeitGeist_tock(scatter_create_mpi_buffer);
          
            ZeitGeist_tick(scatter_create_mpi_buffer);
            //strided_range<Iterator> strided_x(query_outside[proc].begin(),   query_outside[proc].end(), COORD_DIM);
            //strided_range<Iterator> strided_y(query_outside[proc].begin()+1, query_outside[proc].end(), COORD_DIM);
            //strided_range<Iterator> strided_z(query_outside[proc].begin()+2, query_outside[proc].end(), COORD_DIM);
            
            // check the end iterator properly
            strided_range<Iterator> strided_x(query_outside+query_outside_offset[proc],   query_outside+query_outside_offset[proc]+coords_in_proc*COORD_DIM, COORD_DIM);
            strided_range<Iterator> strided_y(query_outside+query_outside_offset[proc]+1, query_outside+query_outside_offset[proc]+coords_in_proc*COORD_DIM, COORD_DIM);
            strided_range<Iterator> strided_z(query_outside+query_outside_offset[proc]+2, query_outside+query_outside_offset[proc]+coords_in_proc*COORD_DIM, COORD_DIM);
            thrust::copy_if(thrust::device, 
                            thrust::make_zip_iterator(thrust::make_tuple(query_points_x_ptr, query_points_y_ptr, query_points_z_ptr)), 
                            thrust::make_zip_iterator(thrust::make_tuple(query_points_x_ptr+N_pts, query_points_y_ptr+N_pts, query_points_z_ptr+N_pts)), 
                            which_proc_ptr, 
                            thrust::make_zip_iterator(thrust::make_tuple(strided_x.begin(), strided_y.begin(), strided_z.begin())), 
                            is_equal(proc));
            ZeitGeist_tock(scatter_create_mpi_buffer);
        }
    }
    timings[3]+=+MPI_Wtime();

    // Now we need to send the query_points that land onto other processor's domain.
    // This done using a sparse alltoallv.
    // Right now each process knows how much data to send to others, but does not know
    // how much data it should receive. This is a necessary information both for the MPI
    // command as well as memory allocation for received data.
    // So we first do an alltoall to get the f_index[proc].size from all processes.

    //for (int proc=0;proc<nprocs;proc++) {
    //  if(!f_index[proc].empty())
    //    f_index_procs_self_sizes[proc]=f_index[proc].size();
    //  else
    //    f_index_procs_self_sizes[proc]=0;
    //}
    
    for (int proc=0;proc<nprocs;proc++) {
        f_index_procs_self_sizes[proc]=num_query_per_proc[proc];
    }
    ZeitGeist_define(scatter_comm_query_size);
    ZeitGeist_tick(scatter_comm_query_size);
    timings[0]+=-MPI_Wtime();
    MPI_Alltoall(f_index_procs_self_sizes,1, MPI_INT,
        f_index_procs_others_sizes,1, MPI_INT,
        c_comm);
    timings[0]+=+MPI_Wtime();
    ZeitGeist_tock(scatter_comm_query_size);


    // Now we need to allocate memory for the receiving buffer of all query
    // points including ours. This is simply done by looping through
    // f_index_procs_others_sizes and adding up all the sizes.
    // Note that we would also need to know the offsets.
    f_index_procs_others_offset[0]=0;
    f_index_procs_self_offset[0]=0;
    for (int proc=0;proc<nprocs;++proc) {
      // The reason we multiply by COORD_DIM is that we have three coordinates per interpolation request
      all_query_points_allocation+=f_index_procs_others_sizes[proc]*COORD_DIM;
      if(proc>0) {
        f_index_procs_others_offset[proc]=f_index_procs_others_offset[proc-1]+f_index_procs_others_sizes[proc-1];
        f_index_procs_self_offset[proc]=f_index_procs_self_offset[proc-1]+f_index_procs_self_sizes[proc-1];
      }
    }
    total_query_points=all_query_points_allocation/COORD_DIM;
    
    //if (this->scatter_baked == false) {
    //  total_query_points_prev = total_query_points;
    //} else {
    //  total_query_points_prev = std::max(total_query_points_prev, total_query_points);
    //}
  
  // This if condition is to allow multiple calls to scatter fucntion with different query points
  // without having to create a new plan
  ZeitGeist_tick(scatter_memalloc);
  if(this->scatter_baked==true) {
    
    if (total_query_points > max_query_points_capacity) {
      PetscPrintf(PETSC_COMM_WORLD, "going to allocate memory again\n");
      // Modify the max memory estimate by increasing neighbour query width by 1 until the requrement is reached
      while (total_query_points > max_query_points_capacity) {
        neighbour_query_width++;
        max_query_points_capacity = get_max_query_allocation(isize, neighbour_query_width);
      }
      hipFree(all_f_cubic_d);
      hipMalloc((void**)&all_f_cubic_d, max_query_points_capacity*sizeof(Real)*data_dof);
      hipFree(all_query_points_d);
      hipMalloc((void**)&all_query_points_d, max_query_points_capacity*COORD_DIM*sizeof(Real) );
      hipFree(xq1);
      hipMalloc((void**)&xq1, max_query_points_capacity*sizeof(Real));
      hipFree(xq2);
      hipMalloc((void**)&xq2, max_query_points_capacity*sizeof(Real));
      hipFree(xq3);
      hipMalloc((void**)&xq3, max_query_points_capacity*sizeof(Real));
    }
      
      // freeing the cuda memory is required everytime scatter is called because the distribution of query points might not be uniform across all GPUs
      //hipFree(all_f_cubic_d);
      //hipMalloc((void**)&all_f_cubic_d, total_query_points*sizeof(Real)*data_dof);
      //
      //hipFree(all_query_points_d);
      //hipMalloc((void**)&all_query_points_d,all_query_points_allocation*sizeof(Real) );
      //
      //hipFree(xq1);
      //hipMalloc((void**)&xq1, total_query_points*sizeof(Real));
      //
      //hipFree(xq2);
      //hipMalloc((void**)&xq2, total_query_points*sizeof(Real));
      //
      //hipFree(xq3);
      //hipMalloc((void**)&xq3, total_query_points*sizeof(Real));
  }
  else {
    // Make an estimate on the number of query points which the current process expects to receive
    // from neighbouring processes
    neighbour_query_width = g_size;
    max_query_points_capacity = get_max_query_allocation(isize, neighbour_query_width);
    while (total_query_points > max_query_points_capacity) {
      neighbour_query_width++;
      max_query_points_capacity = get_max_query_allocation(isize, neighbour_query_width);
    }

    hipMalloc((void**)&all_f_cubic_d, max_query_points_capacity*sizeof(Real)*data_dof);
    hipMalloc((void**)&all_query_points_d, max_query_points_capacity*COORD_DIM*sizeof(Real) );
    hipMalloc((void**)&xq1, max_query_points_capacity*sizeof(Real));
    hipMalloc((void**)&xq2, max_query_points_capacity*sizeof(Real));
    hipMalloc((void**)&xq3, max_query_points_capacity*sizeof(Real));
    
    //hipMalloc((void**)&all_f_cubic_d, total_query_points*sizeof(Real)*data_dof);
    //hipMalloc((void**)&all_query_points_d,all_query_points_allocation*sizeof(Real) );
    //hipMalloc((void**)&xq1, total_query_points*sizeof(Real));
    //hipMalloc((void**)&xq2, total_query_points*sizeof(Real));
    //hipMalloc((void**)&xq3, total_query_points*sizeof(Real));
  }
  ZeitGeist_tock(scatter_memalloc);  

    // Now perform the allotall to send/recv query_points
    ZeitGeist_define(scatter_comm_query_points_sendrcv);
    ZeitGeist_tick(scatter_comm_query_points_sendrcv);
    timings[0]+=-MPI_Wtime();
    int dst_r,dst_s;
    for (int i=0;i<nprocs;++i) {
      dst_r=i;//(procid+i)%nprocs;
      dst_s=i;//(procid-i+nprocs)%nprocs;
      s_request[dst_s]=MPI_REQUEST_NULL;
      request[dst_r]=MPI_REQUEST_NULL;
      //ScalarType* src_ptr = thrust::raw_pointer_cast(query_outside[dst_s].data());
      ScalarType* src_ptr = thrust::raw_pointer_cast(query_outside + query_outside_offset[dst_s]);
      int roffset=f_index_procs_others_offset[dst_r]*COORD_DIM; // notice that COORD_DIM is needed because query_points are 3 times f
      if (i != procid) {
        //int soffset=f_index_procs_self_offset[dst_s]*COORD_DIM;
        if(f_index_procs_others_sizes[dst_r]!=0)
          MPI_Irecv(&all_query_points_d[roffset], f_index_procs_others_sizes[dst_r]*COORD_DIM,MPI_T, dst_r, 0, c_comm, &request[dst_r]);

        //if(!query_outside[dst_s].empty())
        if(num_query_per_proc[dst_s] > 0)
          MPI_Isend(src_ptr, f_index_procs_self_sizes[dst_s]*COORD_DIM, MPI_T, dst_s, 0, c_comm, &s_request[dst_s]);

      } else {
        //if (!query_outside[dst_s].empty())
        if (num_query_per_proc[dst_s] > 0)
          reg::gencpy(&all_query_points_d[roffset], src_ptr, f_index_procs_self_sizes[dst_s]*COORD_DIM*sizeof(ScalarType));
      }
    }
    
    // Wait for all the communication to finish
    MPI_Status ierr;
    for (int proc=0;proc<nprocs;++proc) {
      if(request[proc]!=MPI_REQUEST_NULL)
        MPI_Wait(&request[proc], &ierr);
      if(s_request[proc]!=MPI_REQUEST_NULL)
        MPI_Wait(&s_request[proc], &ierr);
    }
    ZeitGeist_tock(scatter_comm_query_points_sendrcv);
    timings[0]+=+MPI_Wtime();
  
    // Now perform the interpolation on all query points including those that need to
    // be sent to other processors and store them into all_f_cubic
    //free(query_points);
  }

  for(int i=0;i<nprocs;++i){
    MPI_Type_vector(data_dof, f_index_procs_self_sizes[i], N_pts, MPI_T, &rtype[i]);
    MPI_Type_vector(data_dof, f_index_procs_others_sizes[i], total_query_points, MPI_T, &stype[i]);
    MPI_Type_commit(&stype[i]);
    MPI_Type_commit(&rtype[i]);
  }

  int proc_coord[2];
  proc_coord[0] = static_cast<int>(istart[0]/isize[0]);
  proc_coord[1] = static_cast<int>(istart[1]/isize[1]);
  
  // transfer query points "all_query_points" from host to device
  ZeitGeist_define(scatter_query_points_normalize_kernel);
  ZeitGeist_tick(scatter_query_points_normalize_kernel);
  timings[3]+=-MPI_Wtime();
  normalizeQueryPoints(xq1, xq2, xq3, all_query_points_d, total_query_points, isize, N_reg, proc_coord, g_size);
  timings[3]+=+MPI_Wtime();
  ZeitGeist_tock(scatter_query_points_normalize_kernel);
  
  this->scatter_baked=true;
  return;
}


void Interp3_Plan_GPU::test_kernel(Real* f, int nq) {
  test(f, nq);
  return;
}



    


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*
 * Phase 2 of the parallel interpolation: This function must be called after the scatter function is called.
 * It performs local interpolation for all the points that the processor has for itself, as well as the interpolations
 * that it has to send to other processors. After the local interpolation is performed, a sparse
 * alltoall is performed so that all the interpolated results are sent/received.
 *
 */

void Interp3_Plan_GPU::interpolate( Real* ghost_reg_grid_vals_d, // ghost padded regular grid values on GPU
                                    int data_dof,              // degree of freedom for data (vector field=3, scalarfield=1)
                                    int* N_reg,                // size of global grid points 
                                    int* isize,                // size of the local grid owned by the process
                                    int* istart,               // start point of the local grid owned by the process
                                    int* isize_g,              // size of the local grid (including ghost points)
                                    const int nlghost,         // number of local grid points (including ghost points) owned by process
                                    const int N_pts,           // number of local points owned by the process
                                    const int g_size,          // ghost layer width
                                    Real* query_values_d,      // interpolation result on GPU
                                    int* c_dims,               // dimensions of the communicator plan
                                    MPI_Comm c_comm,           // MPI communicator
                                    double * timings,          // time variable to store interpolation time
                                    float *tmp1,               // temporary memory for interpolation prefilter
                                    float* tmp2,               // temporary memory for interpolation prefilter
                                    hipTextureObject_t yi_tex,// texture object for interpolation
                                    int iporder,               // interpolation order
                                    ScalarType* interp_time)   // interpolation time
{
  int nprocs, procid;
  MPI_Comm_rank(c_comm, &procid);
  MPI_Comm_size(c_comm, &nprocs);
  if(this->allocate_baked==false){
    std::cout<<"ERROR Interp3_Plan_GPU interpolate called before calling allocate.\n";
    return;
  }
  if(this->scatter_baked==false){
    std::cout<<"ERROR Interp3_Plan_GPU interpolate called before calling scatter.\n";
    return;
  }

  //PetscSynchronizedPrintf(PETSC_COMM_WORLD, "c_dims = [%d,%d]\n", c_dims[0], c_dims[1]);

#if defined(VERBOSE1) 
  printf("\ng_alloc_max = %zu", g_alloc_max);
  printf("\ndata_dof = %d", data_dof);
  printf("\nisize_g[0] = %d", isize_g[0]);
  printf("\nisize_g[1] = %d", isize_g[1]);
  printf("\nisize_g[2] = %d", isize_g[2]);
  printf("\nipoder = %d", iporder);
  printf("\ntotal_query_points = %d", total_query_points);
  printf("\nnlghost = %d", nlghost);
#endif
  
  // compute the interpolation on the GPU
  ZeitGeist_define(interp_kernel);
  ZeitGeist_tick(interp_kernel);
  timings[1]+=-MPI_Wtime();
  double interp_kernel_time = -MPI_Wtime();
  if (data_dof == 3)
    gpuInterpVec3D(&ghost_reg_grid_vals_d[0*nlghost], 
                   &ghost_reg_grid_vals_d[1*nlghost], 
                   &ghost_reg_grid_vals_d[2*nlghost], 
                   xq1, xq2, xq3, 
                   &all_f_cubic_d[0*total_query_points], 
                   &all_f_cubic_d[1*total_query_points], 
                   &all_f_cubic_d[2*total_query_points], 
                   tmp1, tmp2, isize_g, static_cast<long int>(total_query_points), yi_tex, iporder, interp_time);
  else 
    gpuInterp3D(ghost_reg_grid_vals_d, 
                xq1, xq2, xq3, 
                all_f_cubic_d, 
                tmp1, tmp2, isize_g, static_cast<long int>(total_query_points), yi_tex, 
                iporder, interp_time);
  ZeitGeist_tock(interp_kernel);
  timings[1]+=+MPI_Wtime();
  interp_kernel_time += MPI_Wtime();

#if defined(VERBOSE1) 
  int device;
  hipGetDevice(&device);
  double global_interp_time = 0;
  MPI_Barrier(c_comm);
  MPI_Reduce(&interp_kernel_time, &global_interp_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
  double all_interp_runtimes[nprocs];
  MPI_Gather(&interp_kernel_time, 1, MPI_DOUBLE, all_interp_runtimes, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  PetscPrintf(PETSC_COMM_WORLD, "max =  %0.2E\t", global_interp_time);
  if (procid == 0) {
    PetscPrintf(PETSC_COMM_WORLD, "[");
    for (int i=0; i<nprocs; i++) {
      PetscPrintf(PETSC_COMM_WORLD, "%0.2E," , all_interp_runtimes[i]);
    }
    PetscPrintf(PETSC_COMM_WORLD, "]\n");
  }
#endif

  //std::cout << "interp on device " << device << " took " << interp_kernel_time << "s for " << total_query_points << std::endl;
  
  // Now we have to do an alltoall to distribute the interpolated data from all_f_cubic_d to f_cubic_unordered_d
  ZeitGeist_define(interp_comm_values_sendrcv);
  ZeitGeist_tick(interp_comm_values_sendrcv);
  timings[0]+=-MPI_Wtime();
  int dst_r,dst_s;
  for (int i=0;i<nprocs;++i) {
    dst_r=i;//(procid+i)%nprocs;
    dst_s=i;//(procid-i+nprocs)%nprocs;
    s_request[dst_s]=MPI_REQUEST_NULL;
    request[dst_r]=MPI_REQUEST_NULL;
    // Notice that this is the adjoint of the first comm part
    // because now you are sending others f and receiving your part of f
    int soffset=f_index_procs_others_offset[dst_r];
    int roffset=f_index_procs_self_offset[dst_s];
    if (i != procid) {
      if(f_index_procs_self_sizes[dst_r]!=0)
        MPI_Irecv(&f_cubic_unordered_d[roffset],1,rtype[i], dst_r,
            0, c_comm, &request[dst_r]); 
      if(f_index_procs_others_sizes[dst_s]!=0)
        MPI_Isend(&all_f_cubic_d[soffset],1,stype[i],dst_s,
            0, c_comm, &s_request[dst_s]);
    } else {
      reg::gencpy(&f_cubic_unordered_d[roffset], &all_f_cubic_d[soffset], sizeof(ScalarType)*f_index_procs_self_sizes[i]);
    }
  }

  MPI_Status ierr;
  for (int proc=0;proc<nprocs;++proc){
    if(request[proc]!=MPI_REQUEST_NULL)
      MPI_Wait(&request[proc], &ierr);
    if(s_request[proc]!=MPI_REQUEST_NULL)
      MPI_Wait(&s_request[proc], &ierr);
  }
  
  ZeitGeist_tock(interp_comm_values_sendrcv);
  timings[0]+=+MPI_Wtime();

  
  timings[3]+=-MPI_Wtime();
  ZeitGeist_define(interp_values_copy_kernel);
  ZeitGeist_tick(interp_values_copy_kernel);
  int* f_index_ptr;
  // Now copy back f_cubic_unordered_d to query_values_d in the correct f_index
  for(int dof=0;dof<data_dof;++dof) {
    for(int proc=0;proc<nprocs;++proc) {
      //if(!f_index[proc].empty()) {
      if(num_query_per_proc[proc] > 0) {
          //for (int i=0; i<f_index[proc].size(); ++i) {
          //int ind=f_index[proc][i];
          //query_values_d[ind+dof*N_pts]=f_cubic_unordered_d[f_index_procs_self_offset[proc]+i+dof*N_pts];
          //}
          //PetscSynchronizedPrintf(PETSC_COMM_WORLD, "[%d] proc = %d, f_index[proc].size()=%d\n", procid, proc, f_index[proc].size());
          //PetscSynchronizedFlush(PETSC_COMM_WORLD, PETSC_STDOUT);
          //f_index[proc] = f_index[proc];
          f_index_ptr = thrust::raw_pointer_cast( f_index + f_index_offset[proc] );
          copyQueryValues(&query_values_d[dof*N_pts],
                          &f_cubic_unordered_d[f_index_procs_self_offset[proc]+dof*N_pts], 
                          f_index_ptr, 
                          //f_index[proc].size());
                          num_query_per_proc[proc]);
      }
    }
  }
  ZeitGeist_tock(interp_values_copy_kernel);
  timings[3]+=+MPI_Wtime();

  return;
}
