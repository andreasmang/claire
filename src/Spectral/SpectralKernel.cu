#include "hip/hip_runtime.h"
/*************************************************************************
 *  Copyright (c) 2016.
 *  All rights reserved.
 *  This file is part of the CLAIRE library.
 *
 *  CLAIRE is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  CLAIRE is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with CLAIRE.  If not, see <http://www.gnu.org/licenses/>.
 ************************************************************************/

#ifndef _SPECTRALKERNEL_CPP_
#define _SPECTRALKERNEL_CPP_

#include "SpectralKernel.hpp"
#include "cuda_helper.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "SpectralKernel.txx"

using KernelUtils::SpectralKernelCallGPU;

namespace reg {
  
PetscErrorCode SpectralKernel::LowPassFilter(ComplexType *pXHat, ScalarType pct) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;

  ScalarType l1, l2, l3;
  l1 = static_cast<ScalarType>(nx[0])*0.5*pct;
  l2 = static_cast<ScalarType>(nx[1])*0.5*pct;
  l3 = static_cast<ScalarType>(nx[2])*0.5*pct;
  ierr = SpectralKernelCallGPU<LowPassFilterKernel>(nstart, nx, nl, pXHat, l1, l2, l3, scale); CHKERRQ(ierr);

  PetscFunctionReturn(ierr);
}

PetscErrorCode SpectralKernel::HighPassFilter(ComplexType *pXHat, ScalarType pct) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  ScalarType l1, l2, l3;
  l1 = static_cast<ScalarType>(nx[0])*0.5*pct;
  l2 = static_cast<ScalarType>(nx[1])*0.5*pct;
  l3 = static_cast<ScalarType>(nx[2])*0.5*pct;
  ierr = SpectralKernelCallGPU<HighPassFilterKernel>(nstart, nx, nl, pXHat, l1, l2, l3, scale); CHKERRQ(ierr);
  
  PetscFunctionReturn(ierr);
}

PetscErrorCode SpectralKernel::Scale(ComplexType *pX, ScalarType val) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;

  ierr = SpectralKernelCallGPU<ScaleKernel>(nstart, nx, nl, pX, val); CHKERRQ(ierr);
  
  PetscFunctionReturn(ierr);
}

__global__ void FilterKernel(int3 wave, int3 nl, ComplexType *x, int3 nxc) {
  int i3 = threadIdx.x + blockIdx.x*blockDim.x;
  int i2 = blockIdx.y;
  int i1 = blockIdx.z;
  
  if (i3 < nl.z) {
    wave.x += i1;
    wave.y += i2;
    wave.z += i3;

    int i = i1*nl.y + i2*nl.z + i3;
    
    if (wave.x > nxc.x || wave.y > nxc.y || wave.z > nxc.z) {
      x[i][0] = 0.;
      x[i][1] = 0.;
    }
  }
}

PetscErrorCode SpectralKernel::Restrict(ComplexType *pXc, const ComplexType *pXf, 
                                        const IntType nx_c[3], const IntType osize_c[3], const IntType ostart_c[3]) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  hipMemset(pXc, 0, sizeof(ComplexType)*osize_c[0]*osize_c[1]*osize_c[2]);
  
  size_t pitch_f = nl[2]*sizeof(ComplexType);
  size_t pitch_c = osize_c[2]*sizeof(ComplexType);
  size_t width = (osize_c[2]-1)*sizeof(ComplexType);
  size_t height = osize_c[1]/2;
  
  // width always fits in pencil or slab decomposition;
  
  for (IntType x=0; x<osize_c[0]/2; ++x) {
    size_t offset_c = osize_c[2]*osize_c[1]*x;
    size_t offset_f = nl[2]*nl[1]*x;
    hipMemcpy2DAsync(&pXc[offset_c], pitch_c, const_cast<ComplexType*>(&pXf[offset_f]), pitch_f, width, height, hipMemcpyDeviceToDevice);
    offset_c += osize_c[2]*(osize_c[1] - height);
    offset_f += nl[2]*(nl[1] - height);
    hipMemcpy2DAsync(&pXc[offset_c], pitch_c, const_cast<ComplexType*>(&pXf[offset_f]), pitch_f, width, height, hipMemcpyDeviceToDevice);
  }
  for (IntType x=1; x<=osize_c[0]/2; ++x) {
    size_t offset_c = osize_c[2]*osize_c[1]*(osize_c[0]-x);
    size_t offset_f = nl[2]*nl[1]*(nl[0]-x);
    hipMemcpy2DAsync(&pXc[offset_c], pitch_c, const_cast<ComplexType*>(&pXf[offset_f]), pitch_f, width, height, hipMemcpyDeviceToDevice);
    offset_c += osize_c[2]*(osize_c[1] - height);
    offset_f += nl[2]*(nl[1] - height);
    hipMemcpy2DAsync(&pXc[offset_c], pitch_c, const_cast<ComplexType*>(&pXf[offset_f]), pitch_f, width, height, hipMemcpyDeviceToDevice);
  }
  
  hipDeviceSynchronize();
  
  PetscFunctionReturn(ierr);
}

PetscErrorCode SpectralKernel::Prolong(ComplexType *pXf, const ComplexType *pXc, 
                                       const IntType nx_c[3], const IntType osize_c[3], const IntType ostart_c[3]) {
  PetscErrorCode ierr = 0;
  PetscFunctionBegin;
  
  hipMemset(pXf, 0, sizeof(ComplexType)*nl[0]*nl[1]*nl[2]);
  
  size_t pitch_f = nl[2]*sizeof(ComplexType);
  size_t pitch_c = osize_c[2]*sizeof(ComplexType);
  size_t width = (osize_c[2]-1)*sizeof(ComplexType);
  size_t height = osize_c[1]/2;
  
  for (IntType x=0; x<osize_c[0]/2; ++x) {
    size_t offset_c = osize_c[2]*osize_c[1]*x;
    size_t offset_f = nl[2]*nl[1]*x;
    hipMemcpy2DAsync(&pXf[offset_f], pitch_f, const_cast<ComplexType*>(&pXc[offset_c]), pitch_c, width, height, hipMemcpyDeviceToDevice);
    offset_c += osize_c[2]*(osize_c[1] - height);
    offset_f += nl[2]*(nl[1] - height);
    hipMemcpy2DAsync(&pXf[offset_f], pitch_f, const_cast<ComplexType*>(&pXc[offset_c]), pitch_c, width, height, hipMemcpyDeviceToDevice);
  }
  for (IntType x=1; x<=osize_c[0]/2; ++x) {
    size_t offset_c = osize_c[2]*osize_c[1]*(osize_c[0]-x);
    size_t offset_f = nl[2]*nl[1]*(nl[0]-x);
    hipMemcpy2DAsync(&pXf[offset_f], pitch_f, const_cast<ComplexType*>(&pXc[offset_c]), pitch_c, width, height, hipMemcpyDeviceToDevice);
    offset_c += osize_c[2]*(osize_c[1] - height);
    offset_f += nl[2]*(nl[1] - height);
    hipMemcpy2DAsync(&pXf[offset_f], pitch_f, const_cast<ComplexType*>(&pXc[offset_c]), pitch_c, width, height, hipMemcpyDeviceToDevice);
  }
  
  hipDeviceSynchronize();
  
  PetscFunctionReturn(ierr);
}

} // namepsace reg

#endif
